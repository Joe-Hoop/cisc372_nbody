#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <math.h>
#include "vector.h"
#include "config.h"
#include "planets.h"
#include <hip/hip_runtime.h>
#include "compute.h"

// represents the objects in the system.  Global variables
vector3 *hVel, *d_hVel;
vector3 *hPos, *d_hPos;
double *mass, *d_mass;
vector3 *values, *d_values;
vector3 **accels, **d_accels;
// initHostMemory: Create storage for numObjects entities in our system
// Parameters: numObjects: number of objects to allocate
// Returns: None
// Side Effects: Allocates memory in the hVel, hPos, and mass global variables
void initHostMemory(int numObjects)
{
	hVel = (vector3 *)malloc(sizeof(vector3) * numObjects);
	hPos = (vector3 *)malloc(sizeof(vector3) * numObjects);
	mass = (double *)malloc(sizeof(double) * numObjects);
	values = (vector3 *)malloc(sizeof(vector3) * numObjects * numObjects);
	accels = (vector3 **)malloc(sizeof(vector3 *) * numObjects);
}
void initDeviceMemory(int numObjects)
{
	hipMalloc(&d_hVel, sizeof(vector3) * numObjects);
	hipMalloc(&d_hPos, sizeof(vector3) * numObjects);
	hipMalloc(&d_values, sizeof(vector3) * numObjects * numObjects);
	hipMalloc(&d_accels, sizeof(vector3 *) * numObjects);
	hipMalloc(&d_mass, sizeof(double) * numObjects);
}

// freeHostMemory: Free storage allocated by a previous call to initHostMemory
// Parameters: None
// Returns: None
// Side Effects: Frees the memory allocated to global variables hVel, hPos, and mass.
void freeHostMemory()
{
	free(hVel);
	free(hPos);
	free(mass);
	free(accels);
	free(values);
}

void freeDeviceMemory()
{
	hipFree(d_hVel);
	hipFree(d_hPos);
	hipFree(d_mass);
	hipFree(d_accels);
	hipFree(d_values);
}

// planetFill: Fill the first NUMPLANETS+1 entries of the entity arrays with an estimation
//				of our solar system (Sun+NUMPLANETS)
// Parameters: None
// Returns: None
// Fills the first 8 entries of our system with an estimation of the sun plus our 8 planets.
void planetFill()
{
	int i, j;
	double data[][7] = {SUN, MERCURY, VENUS, EARTH, MARS, JUPITER, SATURN, URANUS, NEPTUNE};
	for (i = 0; i <= NUMPLANETS; i++)
	{
		for (j = 0; j < 3; j++)
		{
			hPos[i][j] = data[i][j];
			hVel[i][j] = data[i][j + 3];
		}
		mass[i] = data[i][6];
	}
}

// randomFill: FIll the rest of the objects in the system randomly starting at some entry in the list
// Parameters: 	start: The index of the first open entry in our system (after planetFill).
//				count: The number of random objects to put into our system
// Returns: None
// Side Effects: Fills count entries in our system starting at index start (0 based)
void randomFill(int start, int count)
{
	int i, j, c = start;
	for (i = start; i < start + count; i++)
	{
		for (j = 0; j < 3; j++)
		{
			hVel[i][j] = (double)rand() / RAND_MAX * MAX_DISTANCE * 2 - MAX_DISTANCE;
			hPos[i][j] = (double)rand() / RAND_MAX * MAX_VELOCITY * 2 - MAX_VELOCITY;
			mass[i] = (double)rand() / RAND_MAX * MAX_MASS;
		}
	}
}

// printSystem: Prints out the entire system to the supplied file
// Parameters: 	handle: A handle to an open file with write access to prnt the data to
// Returns: 		none
// Side Effects: Modifies the file handle by writing to it.
void printSystem(FILE *handle)
{
	int i, j;
	for (i = 0; i < NUMENTITIES; i++)
	{
		fprintf(handle, "pos=(");
		for (j = 0; j < 3; j++)
		{
			fprintf(handle, "%lf,", hPos[i][j]);
		}
		printf("),v=(");
		for (j = 0; j < 3; j++)
		{
			fprintf(handle, "%lf,", hVel[i][j]);
		}
		fprintf(handle, "),m=%lf\n", mass[i]);
	}
}

int main(int argc, char **argv)
{
#define BLOCK_SIZE 16
	clock_t t0 = clock();
	int t_now;
	// srand(time(NULL));
	srand(1234);
	initHostMemory(NUMENTITIES);
	initDeviceMemory(NUMENTITIES);
	planetFill();
	randomFill(NUMPLANETS + 1, NUMASTEROIDS);
// now we have a system.
#ifdef DEBUG
	printSystem(stdout);
#endif
	printf("In the right block\n");
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid((NUMENTITIES + BLOCK_SIZE - 1) / dimBlock.x, (NUMENTITIES + BLOCK_SIZE - 1) / dimBlock.y);

	for (t_now = 0; t_now < DURATION; t_now += INTERVAL)
	{
		hipMemcpy(d_hPos, hPos, sizeof(vector3) * NUMENTITIES, hipMemcpyHostToDevice);
		hipMemcpy(d_hVel, hVel, sizeof(vector3) * NUMENTITIES, hipMemcpyHostToDevice);
		hipMemcpy(d_values, values, sizeof(vector3) * NUMENTITIES * NUMENTITIES, hipMemcpyHostToDevice);
		hipMemcpy(d_accels, accels, sizeof(vector3 *) * NUMENTITIES, hipMemcpyHostToDevice);
		hipMemcpy(d_mass, mass, sizeof(double) * NUMENTITIES, hipMemcpyHostToDevice);

		compute<<<dimGrid, dimBlock>>>(d_values, d_accels, d_hPos, d_hVel, d_mass);
		hipDeviceSynchronize();
		hipMemcpy(hPos, d_hPos, sizeof(vector3) * NUMENTITIES, hipMemcpyDeviceToHost);
		hipMemcpy(hVel, d_hVel, sizeof(vector3) * NUMENTITIES, hipMemcpyDeviceToHost);
		hipMemcpy(values, d_values, sizeof(vector3) * NUMENTITIES * NUMENTITIES, hipMemcpyDeviceToHost);
		hipMemcpy(accels, d_accels, sizeof(vector3 *) * NUMENTITIES, hipMemcpyDeviceToHost);
		hipMemcpy(mass, d_mass, sizeof(double) * NUMENTITIES, hipMemcpyDeviceToHost);
	}

	clock_t t1 = clock() - t0;
#ifdef DEBUG
	printSystem(stdout);
#endif
	printf("This took a total time of %f seconds\n", (double)t1 / CLOCKS_PER_SEC);

	freeHostMemory();
	freeDeviceMemory();
}
